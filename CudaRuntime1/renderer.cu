#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>
#include <time.h>
#include <float.h>
#include "vec3.h"
#include "ray.h"
#include "sphere.h"
#include "hitable_list.h"

#include "renderer.h"

#define DLLEXPORT __declspec(dllexport)

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

DLLEXPORT void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

DLLEXPORT __device__ vec3 color(const ray& r, hitable **world) {
    hit_record rec;
    if ((*world)->hit(r, 0.0, FLT_MAX, rec)) {
        return 0.5f*vec3(rec.normal.x()+1.0f, rec.normal.y()+1.0f, rec.normal.z()+1.0f);
    }
    else {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5f*(unit_direction.y() + 1.0f);
        return (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
    }
}

DLLEXPORT __global__ void render(vec3 *fb, int max_x, int max_y,
                       vec3 lower_left_corner, vec3 horizontal, vec3 vertical, vec3 origin,
                       hitable **world) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    float u = float(i) / float(max_x);
    float v = float(j) / float(max_y);
    ray r(origin, lower_left_corner + u*horizontal + v*vertical);
    fb[pixel_index] = color(r, world);
}

DLLEXPORT __global__ void create_world(hitable **d_list, hitable **d_world) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list)   = new sphere(vec3(0,0,-1), 0.5);
        *(d_list+1) = new sphere(vec3(0,-100.5,-1), 100);
        *d_world    = new hitable_list(d_list,2);
    }
}

DLLEXPORT __global__ void free_world(hitable **d_list, hitable **d_world) {
    delete *(d_list);
    delete *(d_list+1);
    delete *d_world;
}

Renderer::Renderer() {
    nx = 1200;
    ny = 600;
    tx = 8;
    ty = 8;

    std::cerr << "Rendering a " << nx << "x" << ny << " image ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx * ny;
    fb_size = num_pixels * sizeof(vec3);

    // allocate FB
    checkCudaErrors(hipMallocManaged((void**)&fb, fb_size));

    // make our world of hitables
    checkCudaErrors(hipMalloc((void**)&d_list, 2 * sizeof(hitable*)));
    checkCudaErrors(hipMalloc((void**)&d_world, sizeof(hitable*)));
    create_world<<<1,1>>>(d_list, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
}

Renderer::~Renderer() {
    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1, 1>>>(d_list, d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(fb));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
}

void Renderer::render() {
    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx / tx + 1, ny / ty + 1);
    dim3 threads(tx, ty);
    ::render<<<blocks, threads>>>(fb, nx, ny,
        vec3(-2.0, -1.0, -1.0),
        vec3(4.0, 0.0, 0.0),
        vec3(0.0, 2.0, 0.0),
        vec3(0.0, 0.0, 0.0),
        d_world);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";



    // Output FB as Image
    //std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    //for (int j = ny - 1; j >= 0; j--) {
    //    for (int i = 0; i < nx; i++) {
    //        size_t pixel_index = j * nx + i;
    //        int ir = int(255.99 * fb[pixel_index].r());
    //        int ig = int(255.99 * fb[pixel_index].g());
    //        int ib = int(255.99 * fb[pixel_index].b());
    //        std::cout << ir << " " << ig << " " << ib << "\n";
    //    }
    //}
}
